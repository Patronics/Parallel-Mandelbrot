#include "hip/hip_runtime.h"
/*
fractal.cu - Parallel interactive Mandelbrot Fractal Display
based on starting code for CSE 30341 Project 3.
*/
extern "C" {
#include "gfx.h"
}

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <errno.h>
#include <string.h>
#include <complex.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <omp.h>

typedef struct coordSet {
	double xmin;
	double xmax;
	double ymin;
	double ymax;
	int maxiter;
	double xmid;
	double ymid;
} coordSet;


struct colors {
	uint32_t r;
	uint32_t g;
	uint32_t b;
};

void initColors(uint32_t* colorArray)
{
	for(int i = 0; i < 3001; i++)
	{
		colorArray[i] = (255 * i / 3000) << 16;
		colorArray[i] += (((255 * i / 100) & 0xFF) << 8);
		colorArray[i] += ((255 * i / 30) & 0xFF);
	}
	printf("%d\n", colorArray[3000]);
}
/*
Compute the number of iterations at point x, y
n the complex space, up to a maximum of maxiter.
Return the number of iterations at that point.

This example computes the Mandelbrot fractal:
z = z^2 + alpha

Where z is initially zero, and alpha is the location x + iy
in the complex plane.  Note that we are using the "complex"
numeric type in C, which has the special functions cabs()
and cpow() to compute the absolute values and powers of
complex values.
*/

__device__ uint16_t compute_point( double x, double y, uint16_t max )
{
	double z_real = 0;
	double z_imaginary = 0;
	double z_realsquared = 0;
	double z_imaginarysquared = 0;

	uint16_t iter = 0;
	for (iter = 0; iter < max; ++iter) {
		z_imaginary = z_real * z_imaginary;
		z_imaginary = z_imaginary + z_imaginary + y;
		z_real = z_realsquared - z_imaginarysquared + x;
		z_realsquared = z_real * z_real;
		z_imaginarysquared = z_imaginary * z_imaginary;
		if (z_realsquared + z_imaginarysquared >= 4.0) {
			++iter;
			break;
		}
	}

	return iter;
}
uint16_t compute_pointCPU( double x, double y, uint16_t max )
{
        double z_real = 0;
        double z_imaginary = 0;
        double z_realsquared = 0;
        double z_imaginarysquared = 0;

        uint16_t iter = 0;
        for (iter = 0; iter < max; ++iter) {
                z_imaginary = z_real * z_imaginary;
                z_imaginary = z_imaginary + z_imaginary + y;
                z_real = z_realsquared - z_imaginarysquared + x;
                z_realsquared = z_real * z_real;
                z_imaginarysquared = z_imaginary * z_imaginary;
                if (z_realsquared + z_imaginarysquared >= 4.0) {
                        ++iter;
                        break;
                }
        }

        return iter;
}
/*
Compute an entire image, writing each point to the given bitmap.
Scale the image to the range (xmin-xmax,ymin-ymax).
*/

__global__ void compute_image(double xmin, double ymin, int maxiter, int width, int height, uint32_t* colorsArray, uint32_t* cols, double stepFactorX, double stepFactorY)
{
	//double xmin=coords->xmin;
	//double xmax=coords->xmax;
	//double ymin=coords->ymin;
	//double ymax=coords->ymax;
	//int maxiter=coords->maxiter;

    uint32_t my_i = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t my_j = blockDim.y * blockIdx.y + threadIdx.y;
	//int total_threads = gridDim.x * blockDim.x;
	//int total_threads1 = gridDim.y * blockDim.y;
	if(my_i < width && my_j < height) {	
		
    double x = xmin + my_i*stepFactorX;
	double y = ymin + my_j*stepFactorY;

    uint16_t iter = compute_point(x,y,maxiter);
    //colorsSet[my_i+width*my_j].r = 255 * iter / maxiter;
	//colorsSet[my_i+width*my_j].g = 255 * iter / (maxiter/30);
	//colorsSet[my_i+width*my_j].b = 255 * iter / (maxiter/100);
    	//colorsSet[my_i+width*my_j].r = (colorsArray[iter] & 0xFF0000) >> 16;
	//colorsSet[my_i+width*my_j].b = (colorsArray[iter] & 0xFF00) >> 8;
	//colorsSet[my_i+width*my_j].g = colorsArray[iter] & 0xFF;
	cols[my_i+width*my_j] = colorsArray[iter];
	}
}

void compute_imageCPU(coordSet* coords, int width, int height, struct colors *colorsSet, uint32_t* colorsArray, uint32_t* cols)
{
        double xmin=coords->xmin;
        double xmax=coords->xmax;
        double ymin=coords->ymin;
        double ymax=coords->ymax;
        int maxiter=coords->maxiter;
	double start_time = omp_get_wtime();
    //int my_i = blockDim.x * blockIdx.x + threadIdx.x;
    //int my_j = blockDim.y * blockIdx.y + threadIdx.y;
        //int total_threads = gridDim.x * blockDim.x;
        //int total_threads1 = gridDim.y * blockDim.y;
	#pragma omp parallel for schedule(dynamic)
        for(int i = height*0.6; i < height; ++i) {

		#pragma omp parallel for schedule(dynamic)
		for(int j = 0; j < width; ++j) {

    double x = xmin + j*(xmax-xmin)/width;
        double y = ymin + i*(ymax-ymin)/height;

    uint16_t iter = compute_pointCPU(x,y,maxiter);
    //colorsSet[i*width+j].r = 255 * iter / maxiter;
        //colorsSet[i*width+j].g = 255 * iter / (maxiter/30);
        //colorsSet[i*width+j].b = 255 * iter / (maxiter/100);
	//colorsSet[i*width+j].r = (colorsArray[iter] & 0xFF0000) >> 16;
        //colorsSet[i*width+j].b = (colorsArray[iter] & 0xFF00) >> 8;
        //colorsSet[i*width+j].g = colorsArray[iter] & 0xFF;
    	cols[i*width+j] = colorsArray[iter];
        }
	}
	double end_time = omp_get_wtime();
        printf("%.5f\n", end_time - start_time);
}


void draw_point(int i, int j, struct colors c,uint32_t* cols)
{
	gfx_color(c.r, c.g, c.b);
	// Plot the point on the screen.
	gfx_point(j, i);
}

void setMidpoints(coordSet* coords){
	coords->xmid = (coords->xmin+coords->xmax)/2;
	coords->ymid = (coords->ymin+coords->ymax)/2;

}

void reDraw(coordSet* coords){
    //int width = gfx_xsize();
	//int height = gfx_ysize();
	int width = 18000;
	int height = 18000;
    int n = width * height;
	
	//TODO bigger blocks are likely faster
	#define BLOCK_SIZE 16
    	double xMin = coords->xmin;
	double yMin = coords->ymin;
	double xStep = (coords->xmax-coords->xmin)/width;
	double yStep = (coords->ymax-coords->ymin)/height;
	double maxIter = coords->maxiter;
	dim3 dimBlock(16, 16); // so your threads are BLOCK_SIZE*BLOCK_SIZE, 256 in this case
	dim3 dimGrid(width/BLOCK_SIZE, height/BLOCK_SIZE); // 1*1 blocks in a grid
	uint32_t* colorsArray = (uint32_t*) malloc(3001 * sizeof(uint32_t));
	//struct colors* colorsSet;
	//struct colors* c = (struct colors*)malloc(n * sizeof(struct colors));
	uint32_t* cols = (uint32_t*)malloc(n*sizeof(uint32_t));
	uint32_t* cudacols;
	hipMalloc(&cudacols, n * sizeof(uint32_t));
	initColors(colorsArray);
	//hipMalloc(&colorsSet, n * sizeof(struct colors));
	//coordSet* cudaCoords;
	//hipMalloc(&cudaCoords, sizeof(coordSet));
	uint32_t* cuda_array;
	hipMalloc(&cuda_array, 3001 * sizeof(uint32_t));
	// Show the configuration, just in case you want to recreate it.
	//printf("coordinates: %lf %lf %lf %lf\n",coords->xmin,coords->xmax,coords->ymin,coords->ymax);
	// Display the fractal image

	struct timespec startTime, endTime;
	double runTime;
	//clock_gettime(CLOCK_MONOTONIC, &startTime);
	// this is not the actual block size and thread count
	hipError_t whatever = hipMemcpy(cuda_array, colorsArray, 3001 * sizeof(uint32_t), hipMemcpyHostToDevice);
	//hipError_t err = hipMemcpy(cudaCoords, coords,sizeof(coordSet), hipMemcpyHostToDevice);
	//if (err != hipSuccess) printf("%s memcpy0 coords\n", hipGetErrorString(err));
	//err = hipMemcpy(colorsSet, c, n * sizeof(struct colors), hipMemcpyHostToDevice);
	hipMemcpy(cudacols,cols,n*sizeof(uint32_t),hipMemcpyHostToDevice);
	//if (err != hipSuccess) printf("%s memcpy1\n", hipGetErrorString(err));
	double start = omp_get_wtime();
	compute_image <<<dimGrid, dimBlock>>>(xMin,yMin,maxIter, width, height,cuda_array,cudacols,xStep,yStep);
	//double end = omp_get_wtime();
	//compute_imageCPU(coords, width, height, c,colorsArray,cols);
	//err = hipDeviceSynchronize();
	//double end = omp_get_wtime();
	//if (err != hipSuccess) printf("%s synch\n", hipGetErrorString(err));
	//err = hipMemcpy(c, colorsSet, n * sizeof(struct colors), hipMemcpyDeviceToHost);
	hipError_t err = hipMemcpy(cols,cudacols, n * sizeof(uint32_t), hipMemcpyDeviceToHost);
	//if (err != hipSuccess) printf("%s memcpy2\n", hipGetErrorString(err));
	//err = hipDeviceSynchronize();
	
	//clock_gettime(CLOCK_MONOTONIC, &endTime);
	//runTime = difftime(endTime.tv_sec, startTime.tv_sec)+((endTime.tv_nsec-startTime.tv_nsec)/1e9);
	//fprintf(stderr, "calculating frame took %lf seconds\n", runTime);
	double end = omp_get_wtime();
	printf("time: %lf\n", end - start);	
	for (int i = 0; i < height; i++)
		for (int j = 0; j < width; j++){
			//c[i * width + j].r=j;
			uint32_t r = (cols[i*width+j] & 0xFF0000) >> 16;
		       	uint32_t g = (cols[i*width+j] & 0xFF00) >> 8;
			uint32_t b = (cols[i*width+j] & 0xFF);	
			        gfx_color(r,g,b);
        			// Plot the point on the screen.
        			gfx_point(j, i);
		}
	
	//clock_gettime(CLOCK_MONOTONIC, &endTime);
	//runTime = difftime(endTime.tv_sec, startTime.tv_sec)+((endTime.tv_nsec-startTime.tv_nsec)/1e9);
	//fprintf(stderr, "\ncalculating and rendering frame took %lf seconds\n", runTime);

	//free(c);
	//hipFree(colorsSet);
}


void zoomIn(coordSet* coords,double extent){
	setMidpoints(coords);
	double width = coords->xmid-coords->xmin;
	double height = coords->ymid-coords->ymin;
	coords->xmax=coords->xmid+(width/extent);
	coords->xmin=coords->xmid-(width/extent);
	coords->ymax=coords->ymid+(height/extent);
	coords->ymin=coords->ymid-(height/extent);
	setMidpoints(coords);
	reDraw(coords);
}

void zoomOut(coordSet* coords, double extent){
	setMidpoints(coords);
	double width = coords->xmid-coords->xmin;
	double height = coords->ymid-coords->ymin;
	coords->xmax=coords->xmid+(width*extent);
	coords->xmin=coords->xmid-(width*extent);
	coords->ymax=coords->ymid+(height*extent);
	coords->ymin=coords->ymid-(height*extent);
	setMidpoints(coords);
	reDraw(coords);
}

void shiftFrame(coordSet* coords, double xShift, double yShift){
	setMidpoints(coords);
	double width = coords->xmax-coords->xmin;
	double height = coords->ymax-coords->ymin;
	coords->xmax+=xShift*width;
	coords->xmin+=xShift*width;
	coords->ymax+=yShift*height;
	coords->ymin+=yShift*height;
	setMidpoints(coords);
	reDraw(coords);
}


//accidentally discovered, mirrors coords, may be useful
void reflect(coordSet* coords){
	setMidpoints(coords);
	coords->xmax=coords->xmax+(coords->xmid-coords->xmax)*2;
	coords->xmin=coords->xmin+(coords->xmid-coords->xmin)*2;
	coords->ymax=coords->ymax+(coords->ymid-coords->ymax)*2;
	coords->ymin=coords->ymin+(coords->ymid-coords->ymin)*2;
	setMidpoints(coords);
	reDraw(coords);
}


int main( int argc, char *argv[] ){
	// The initial boundaries of the fractal image in x,y space.
	const double xminDefault = -2.0;
	const double xmaxDefault = 1.0;
	const double yminDefault = -1.5;
	const double ymaxDefault=  1.5;
	// Maximum number of iterations to compute.
	// Higher values take longer but have more detail.
	const int maxiterDefault = 3000; //default 500
	
	coordSet* dispCoords = (coordSet*)malloc(sizeof(coordSet));
	
	if(argv[1] && argv[2] && argv[3] && argv[4] && argv[5]){
		dispCoords->xmin = atof(argv[1]);
		dispCoords->xmax = atof(argv[2]);
		dispCoords->ymin = atof(argv[3]);
		dispCoords->ymax = atof(argv[4]);
		dispCoords->maxiter = atoi(argv[5]);
		setMidpoints(dispCoords);
	}else{
		dispCoords->xmin=xminDefault;
		dispCoords->xmax=xmaxDefault;
		dispCoords->ymin=yminDefault;
		dispCoords->ymax=ymaxDefault;
		dispCoords->maxiter=maxiterDefault;
		setMidpoints(dispCoords);
	}


	// Open a new window.
	gfx_open(640,640,"Mandelbrot Fractal");


	// Fill it with a dark blue initially.
	gfx_clear_color(0,0,255);
	gfx_clear();

	//draw intial position
	reDraw(dispCoords);


	while(1) {
		// Wait for a key or mouse click.
		int c = gfx_wait();
		printf("got character %c\n",c);
		// Quit if q is pressed.
		switch(c){
		case 'q':
			free(dispCoords);
			exit(0);
		//reset default position
		case 'r':
			dispCoords->xmin=xminDefault;
			dispCoords->xmax=xmaxDefault;
			dispCoords->ymin=yminDefault;
			dispCoords->ymax=ymaxDefault;
			setMidpoints(dispCoords);
			reDraw(dispCoords);
			break;
		//Reflect the view (mirroring it)
		case 'R':
			reflect(dispCoords);
			break;
		//zoom in/out with i/o (or smoothly with I/O)
		case 'i':
			printf("zooming in\n");
			zoomIn(dispCoords, 2);
			break;
		case 'o':
			printf("zooming out\n");
			zoomOut(dispCoords, 2);
			break;
		case 'I':
			printf("zooming in slightly\n");
			zoomIn(dispCoords, 1.25);
			break;
		case 'O':
			printf("zooming out slightly\n");
			zoomOut(dispCoords, 1.25);
			break;
		//pan with wasd (or smoothly with WASD)
		case 'w':
			shiftFrame(dispCoords, 0, -0.5);
			break;
		case 'W':
			shiftFrame(dispCoords, 0, -0.25);
			break;
		case 's':
			shiftFrame(dispCoords, 0, 0.5);
			break;
		case 'S':
			shiftFrame(dispCoords, 0, 0.25);
			break;
		case 'a':
			shiftFrame(dispCoords, -0.5, 0);
			break;
		case 'A':
			shiftFrame(dispCoords, -0.25, 0);
			break;
		case 'd':
			shiftFrame(dispCoords, 0.5, 0);
			break;
		case 'D':
			shiftFrame(dispCoords, 0.25, 0);
			break;
		}
//		} else if(c=='q'){
	}

	return 0;
}
