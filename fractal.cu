#include "hip/hip_runtime.h"
/*
fractal.cu - Parallel interactive Mandelbrot Fractal Display
based on starting code for CSE 30341 Project 3.
*/
extern "C" {
#include "gfx.h"
}

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <errno.h>
#include <string.h>
#include <complex.h>
#include <time.h>

#include <hip/hip_runtime.h>

typedef struct coordSet {
	double xmin;
	double xmax;
	double ymin;
	double ymax;
	int maxiter;
	double xmid;
	double ymid;
} coordSet;


struct colorss {
	int r;
	int g;
	int b;
};

/*
Compute the number of iterations at point x, y
in the complex space, up to a maximum of maxiter.
Return the number of iterations at that point.

This example computes the Mandelbrot fractal:
z = z^2 + alpha

Where z is initially zero, and alpha is the location x + iy
in the complex plane.  Note that we are using the "complex"
numeric type in C, which has the special functions cabs()
and cpow() to compute the absolute values and powers of
complex values.
*/

__device__ int compute_point( double x, double y, int max )
{
	double z_real = 0;
	double z_imaginary = 0;
	double z_realsquared = 0;
	double z_imaginarysquared = 0;

	int iter = 0;
	for (iter = 0; iter < max; iter++) {
		z_imaginary = z_real * z_imaginary;
		z_imaginary = z_imaginary + z_imaginary + y;
		z_real = z_realsquared - z_imaginarysquared + x;
		z_realsquared = z_real * z_real;
		z_imaginarysquared = z_imaginary * z_imaginary;
		if (z_realsquared + z_imaginarysquared >= 4.0) {
			iter++;
			break;
		}
	}

	return iter;
}

/*
Compute an entire image, writing each point to the given bitmap.
Scale the image to the range (xmin-xmax,ymin-ymax).
*/

__global__ void compute_image(coordSet* coords, int width, int height, struct colorss *colorsset)
{
	double xmin=coords->xmin;
	double xmax=coords->xmax;
	double ymin=coords->ymin;
	double ymax=coords->ymax;
	int maxiter=coords->maxiter;

    int my_i = blockDim.x * blockIdx.x + threadIdx.x;
    int my_j = blockDim.y * blockIdx.y + threadIdx.y;

    double x = xmin + my_i*(xmax-xmin)/width;
	double y = ymin + my_j*(ymax-ymin)/height;

    int iter = 0;
    //iter = compute_point(x,y,maxiter);
	for(int i=0; i<100000; i++){
	    colorsset[i].r = 255;//* iter / maxiter;
		colorsset[my_i].g = 255 * iter / (maxiter/30);
		colorsset[my_i].b = 255 * iter / (maxiter/100);
	}

}

void draw_point(int i, int j, struct colorss c)
{
	gfx_color(c.r, c.g, c.b);
	// Plot the point on the screen.
	gfx_point(j, i);
}

void setMidpoints(coordSet* coords){
	coords->xmid = (coords->xmin+coords->xmax)/2;
	coords->ymid = (coords->ymin+coords->ymax)/2;

}

void reDraw(coordSet* coords){
    int width = gfx_xsize();
	int height = gfx_ysize();

    int n = width * height;

	struct colorss* colorsset;
	struct colorss* c = (struct colorss*)malloc(n * sizeof(struct colorss));
	hipMalloc(&colorsset, n * sizeof(struct colorss));
	coordSet* cudaCoords;
	hipMalloc(&cudaCoords, sizeof(coordSet));
	// Show the configuration, just in case you want to recreate it.
	printf("coordinates: %lf %lf %lf %lf\n",coords->xmin,coords->xmax,coords->ymin,coords->ymax);
	// Display the fractal image

	struct timespec startTime, endTime;
	double runTime;
	clock_gettime(CLOCK_MONOTONIC, &startTime);
	// this is not the actual block size and thread count
	hipError_t err = hipMemcpy(cudaCoords, coords,sizeof(coordSet), hipMemcpyHostToDevice);
	if (err != hipSuccess) printf("%s memcpy0 coords\n", hipGetErrorString(err));
	err = hipMemcpy(colorsset, c, n * sizeof(struct colorss), hipMemcpyHostToDevice);
	if (err != hipSuccess) printf("%s memcpy1\n", hipGetErrorString(err));
	compute_image <<<1, 1>>>(cudaCoords, width, height, colorsset);
	err = hipDeviceSynchronize();
	printf("%s synch\n", hipGetErrorString(err));
	err = hipMemcpy(c, colorsset, n * sizeof(struct colorss), hipMemcpyDeviceToHost);
	if (err != hipSuccess) printf("%s memcpy2\n", hipGetErrorString(err));
	err = hipDeviceSynchronize();
	//if (err != hipSuccess)
	
	for (int i = 0; i < height; i++)
		for (int j = 0; j < width; j++){
			//c[i * width + j].r=j;
			draw_point(i, j, c[i * width + j]);
		}
	clock_gettime(CLOCK_MONOTONIC, &endTime);
	runTime = difftime(endTime.tv_sec, startTime.tv_sec)+((endTime.tv_nsec-startTime.tv_nsec)/1e9);
	fprintf(stderr, "\nrendering frame took %lf seconds\n", runTime);

	free(c);
	hipFree(colorsset);
}


void zoomIn(coordSet* coords,double extent){
	setMidpoints(coords);
	double width = coords->xmid-coords->xmin;
	double height = coords->ymid-coords->ymin;
	coords->xmax=coords->xmid+(width/extent);
	coords->xmin=coords->xmid-(width/extent);
	coords->ymax=coords->ymid+(height/extent);
	coords->ymin=coords->ymid-(height/extent);
	setMidpoints(coords);
	reDraw(coords);
}

void zoomOut(coordSet* coords, double extent){
	setMidpoints(coords);
	double width = coords->xmid-coords->xmin;
	double height = coords->ymid-coords->ymin;
	coords->xmax=coords->xmid+(width*extent);
	coords->xmin=coords->xmid-(width*extent);
	coords->ymax=coords->ymid+(height*extent);
	coords->ymin=coords->ymid-(height*extent);
	setMidpoints(coords);
	reDraw(coords);
}

void shiftFrame(coordSet* coords, double xShift, double yShift){
	setMidpoints(coords);
	double width = coords->xmax-coords->xmin;
	double height = coords->ymax-coords->ymin;
	coords->xmax+=xShift*width;
	coords->xmin+=xShift*width;
	coords->ymax+=yShift*height;
	coords->ymin+=yShift*height;
	setMidpoints(coords);
	reDraw(coords);
}


//accidentally discovered, mirrors coords, may be useful
void reflect(coordSet* coords){
	setMidpoints(coords);
	coords->xmax=coords->xmax+(coords->xmid-coords->xmax)*2;
	coords->xmin=coords->xmin+(coords->xmid-coords->xmin)*2;
	coords->ymax=coords->ymax+(coords->ymid-coords->ymax)*2;
	coords->ymin=coords->ymin+(coords->ymid-coords->ymin)*2;
	setMidpoints(coords);
	reDraw(coords);
}


int main( int argc, char *argv[] ){
	// The initial boundaries of the fractal image in x,y space.
	const double xminDefault = -1.5;
	const double xmaxDefault = 0.5;
	const double yminDefault = -1.0;
	const double ymaxDefault=  1.0;
	// Maximum number of iterations to compute.
	// Higher values take longer but have more detail.
	const int maxiterDefault = 3000; //default 500
	
	coordSet* dispCoords = (coordSet*)malloc(sizeof(coordSet));
	
	if(argv[1] && argv[2] && argv[3] && argv[4] && argv[5]){
		dispCoords->xmin = atof(argv[1]);
		dispCoords->xmax = atof(argv[2]);
		dispCoords->ymin = atof(argv[3]);
		dispCoords->ymax = atof(argv[4]);
		dispCoords->maxiter = atoi(argv[5]);
		setMidpoints(dispCoords);
	}else{
		dispCoords->xmin=xminDefault;
		dispCoords->xmax=xmaxDefault;
		dispCoords->ymin=yminDefault;
		dispCoords->ymax=ymaxDefault;
		dispCoords->maxiter=maxiterDefault;
		setMidpoints(dispCoords);
	}


	// Open a new window.
	gfx_open(640,480,"Mandelbrot Fractal");


	// Fill it with a dark blue initially.
	gfx_clear_color(0,0,255);
	gfx_clear();

	//draw intial position
	reDraw(dispCoords);


	while(1) {
		// Wait for a key or mouse click.
		int c = gfx_wait();
		printf("got character %c\n",c);
		// Quit if q is pressed.
		switch(c){
		case 'q':
			free(dispCoords);
			exit(0);
		//reset default position
		case 'r':
			dispCoords->xmin=xminDefault;
			dispCoords->xmax=xmaxDefault;
			dispCoords->ymin=yminDefault;
			dispCoords->ymax=ymaxDefault;
			setMidpoints(dispCoords);
			reDraw(dispCoords);
			break;
		//Reflect the view (mirroring it)
		case 'R':
			reflect(dispCoords);
			break;
		//zoom in/out with i/o (or smoothly with I/O)
		case 'i':
			printf("zooming in\n");
			zoomIn(dispCoords, 2);
			break;
		case 'o':
			printf("zooming out\n");
			zoomOut(dispCoords, 2);
			break;
		case 'I':
			printf("zooming in slightly\n");
			zoomIn(dispCoords, 1.25);
			break;
		case 'O':
			printf("zooming out slightly\n");
			zoomOut(dispCoords, 1.25);
			break;
		//pan with wasd (or smoothly with WASD)
		case 'w':
			shiftFrame(dispCoords, 0, -0.5);
			break;
		case 'W':
			shiftFrame(dispCoords, 0, -0.25);
			break;
		case 's':
			shiftFrame(dispCoords, 0, 0.5);
			break;
		case 'S':
			shiftFrame(dispCoords, 0, 0.25);
			break;
		case 'a':
			shiftFrame(dispCoords, -0.5, 0);
			break;
		case 'A':
			shiftFrame(dispCoords, -0.25, 0);
			break;
		case 'd':
			shiftFrame(dispCoords, 0.5, 0);
			break;
		case 'D':
			shiftFrame(dispCoords, 0.25, 0);
			break;
		}
//		} else if(c=='q'){
	}

	return 0;
}
